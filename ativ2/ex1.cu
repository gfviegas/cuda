#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

double cpuSecond() {
  struct timeval tp;
  gettimeofday(&tp, NULL);
  return ((double) tp.tv_sec + (double) tp.tv_usec * 1.e-6);
}

#define CHECK(call)                                         \
{                                                           \
  const hipError_t error = call;                           \
  if (error != hipSuccess) {                               \
    fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);  \
    fprintf(stderr, "code: %d, reason: %s\n", error,        \
    hipGetErrorString(error));                             \
    exit(1);                                                \
  }                                                         \
}

void initialData(float *ip, int size){
  time_t t;
  srand((unsigned int) time (&t));

  for (int i = 0; i < size; i++){
    ip[i] = (float)(rand() & 0xFF) / 10.0f;
  }
}

void sumArraysOnHost(float *A, float *B, float *C, const int N) {
  for (int idx = 0; idx < N; idx++) {
    C[idx] = A[idx] + B[idx];
  }
}

__global__ void sumArraysOnGpu(float *A, float *B, float *C, const int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < N) {
    C[i] = A[i] + B[i];
  }
}


int main(int argc, char**argv) {
  // Configura tamanho dos vetores
  int nElem = 177 * 1.e6;
  printf("Tamanho dos vetores: %d\n", nElem);

  // Alocando memoria na CPU
  size_t nBytes = nElem * sizeof(float);

  float *h_A, *h_B, *hostRef, *gpuRef;
  h_A = (float *) malloc(nBytes);
  h_B = (float *) malloc(nBytes);
  hostRef = (float *) malloc(nBytes);
  gpuRef = (float *) malloc(nBytes);

  initialData(h_A, nElem);
  initialData(h_B, nElem);

  // Alocando memoria global (GPU)
  float *d_A, *d_B, *d_C;
  CHECK(hipMalloc((float **)&d_A, nBytes));
  CHECK(hipMalloc((float **)&d_B, nBytes));
  CHECK(hipMalloc((float **)&d_C, nBytes));

  // Transferindo dados da CPU pra GPU
  CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
  // CHECK(cudaMemcpy(d_A, h_A, nBytes, cudaMemcpyHostToDevice));

  // Invocando o Kernel na CPU
  int iLen = 512;
  dim3 block(iLen);
  dim3 grid((nElem + block.x - 1) / block.x);
  sumArraysOnGpu<<<grid, block>>>(d_A, d_B, d_C, nElem);

  // Copia os resultados do Kernel de volta pra CPU
  CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

  // Libera memoria da GPU
  CHECK(hipFree(d_A));
  CHECK(hipFree(d_B));
  CHECK(hipFree(d_C));

  // Libera memória da CPU
  free(h_A);
  free(h_B);
  free(hostRef);
  free(gpuRef);

  hipDeviceReset();
  return 0;
}
