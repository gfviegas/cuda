#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

__global__ void sumArraysOnGpu(float *A, float *B, float *C, int fatorUnroll) {
  unsigned int idx = blockIdx.x * blockDim.x * fatorUnroll + threadIdx.x;

  for (int i = 1; i <= fatorUnroll; i++) {
    int index = idx + fatorUnroll;
    C[index] = A[index] + B[index];
  }
}

void initialData(float *ip, int size){
  // generate different seed for random number
  time_t t;
  srand((unsigned int) time (&t) - ip[0]);

  for (int i=0; i<size; i++){
    ip[i] = (float)(rand() & 0xFF) / 10.0f;
  }
}

void linearData(float *input, int size) {
  for (int i = 0; i < size; i++) {
    input[i] = i + (size / (1024 * 1e3));
  }
}

int main(int argc, char **argv){
  int expoente = atoi(argv[1]); // Primeiro argumento é o expoente onde 2^X = tamanho do elemento
  int threads = atoi(argv[2]); // Segundo argumento é o numero de threads
  int fatorUnroll = atoi(argv[3]); // Terceiro argumento é o fator de unroll

  size_t nBytes = (2 << (expoente + 1)) / sizeof(float);
  int nElem = nBytes / sizeof(float);

  float *h_A, *h_B, *h_C;
  h_A = (float *)malloc(nBytes);
  h_B = (float *)malloc(nBytes);
  h_C = (float *)malloc(nBytes);

  initialData(h_A, nElem);
  linearData(h_B, nElem);

  printf("Quantidade de elementos: %d \n Quantidade de MB: %lu MB\n\n", nElem, (nBytes / (1024*1024)));

  float *d_A, *d_B, *d_C;
  hipMalloc((float**)&d_A, nBytes);
  hipMalloc((float**)&d_B, nBytes);
  hipMalloc((float**)&d_C, nBytes);

  // Use cudaMemcpy to transfer the data from the host memory to the GPU global memory with the
  // parameter cudaMemcpyHostToDevice specifying the transfer direction.
  hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

  sumArraysOnGpu<<<(nElem / fatorUnroll) / threads, threads>>>(d_A, d_B, d_C, fatorUnroll);

  hipMemcpy(h_C, d_C, nBytes, hipMemcpyDeviceToHost);

  free(h_A);
  free(h_B);
  free(h_C);

  // use cudaFree to release the memory used on the GPU
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipDeviceReset();

  return (0);
}
