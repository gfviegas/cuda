#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void sumArraysOnGpu(float *A, float *B, float *C){
  int idx = threadIdx.x;
  C[idx] = A[idx] + B[idx];
}

__global__ void mathOperationsOnGPU(float *A, float *B, float *C, int operations) {
  int idx = threadIdx.x;
  float result;

  for (int i = 0; i < operations; i++) {
    int r = i % 6;

    switch (r) {
      case 0:
        result += B[idx];
        break;
      case 1:
      result -= A[idx];
        break;
      case 2:
        result += 9;
        break;
      case 3:
        result -= 9203.34;
        break;
      case 4:
        result *= 0.2;
        break;
      case 5:
        result -= (A[idx] / 1024);
        break;
    }
  }

  C[idx] = result;
}


void initialData(float *ip, int size){
  // generate different seed for random number
  time_t t;
  srand((unsigned int) time (&t));

  for (int i=0; i<size; i++){
    ip[i] = (float)(rand() & 0xFF) / 10.0f;
  }
}

void linearData(float *input, int size) {
  for (int i = 0; i < size; i++) {
    input[i] = i + (size / (1024 * 1e3));
  }
}

int main(int argc, char **argv) {
  int expoente = atoi(argv[1]); // Primeiro argumento é o expoente onde 2^X = tamanho do elemento
  int blocks = atoi(argv[2]); // Primeiro argumento é a quantidade de blocos
  int operations = atoi(argv[3]); // Segundo argumento é a quantidade de operações matemáticas por thread

  srand(time(NULL));

  size_t nBytes = (2 << (expoente + 1)) / sizeof(float);
  int nElem = nBytes / sizeof(float);

  float *h_A, *h_B, *h_C, *result;
  h_A = (float *)malloc(nBytes);
  h_B = (float *)malloc(nBytes);
  h_C = (float *)malloc(nBytes);
  result = (float *)malloc(nBytes);

  initialData(h_A, nElem);
  linearData(h_B, nElem);

  printf("Quantidade de elementos: %d \n Quantidade de MB: %lu MB, Quantidade de operações: %d\n\n", nElem, (nBytes / (1024*1024)), operations);

  float *d_A, *d_B, *d_C;
  hipMalloc((float**)&d_A, nBytes);
  hipMalloc((float**)&d_B, nBytes);
  hipMalloc((float**)&d_C, nBytes);

  // Use cudaMemcpy to transfer the data from the host memory to the GPU global memory with the
  // parameter cudaMemcpyHostToDevice specifying the transfer direction.
  hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

  mathOperationsOnGPU<<<blocks, 1024>>>(d_A, d_B, d_C, operations);
  // sumArraysOnGpu<<<1, nElem>>>(d_A, d_B, d_C);
  hipMemcpy(h_C, d_C, nBytes, hipMemcpyDeviceToHost);

  free(h_A);
  free(h_B);
  free(h_C);
  free(result);

  // use cudaFree to release the memory used on the GPU
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipDeviceReset();

  return (0);
}
