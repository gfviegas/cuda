#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

double cpuSecond() {
  struct timeval tp;
  gettimeofday(&tp, NULL);
  return ((double) tp.tv_sec + (double) tp.tv_usec * 1.e-6);
}

#define CHECK(call)                                         \
{                                                           \
  const hipError_t error = call;                           \
  if (error != hipSuccess) {                               \
    fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);  \
    fprintf(stderr, "code: %d, reason: %s\n", error,        \
    hipGetErrorString(error));                             \
    exit(1);                                                \
  }                                                         \
}

void identityData(int* I, int nElem) {
  for (int i = 0; i < nElem; i++) {
    I[i] = i;
  }
}

void initialData(float *ip, int size){
  time_t t;
  srand((unsigned int) time (&t));

  for (int i = 0; i < size; i++){
    ip[i] = (float)(rand() & 0xFF) / 10.0f;
  }
}

void initialDataInt(int *ip, int size){
  time_t t;
  srand((unsigned int) time (&t));

  for (int i = 0; i < size; i++){
    ip[i] = floor((rand() & 0xFF) / 10.0f);
  }
}

void sumArraysOnHost(float *A, float *B, float *C, const int N) {
  for (int idx = 0; idx < N; idx++) {
    C[idx] = A[idx] + B[idx];
  }
}

__global__ void sumArraysOnGpu(float *A, float *B, float *C, int* I, int* R, int strike, const int N) {
    __shared__ float smem[512];

    // número de conflitos
    int conflicts = 8;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        smem[threadIdx.x] += i;
        C[i] = A[i] + B[i] + smem[(threadIdx.x * conflicts) % blockDim.x];
    }
}


int main(int argc, char**argv) {
  // Configura tamanho dos vetores
  int nElem = 100 * 1.e6;
  int strike = 1;

  // Alocando memoria na CPU
  size_t nBytes = nElem * sizeof(float);

  float *h_A, *h_B, *hostRef, *gpuRef;
  int *R, *I;

  h_A = (float *) malloc(nBytes);
  h_B = (float *) malloc(nBytes);
  R = (int *) malloc(nBytes);
  I = (int *) malloc(nBytes);

  hostRef = (float *) malloc(nBytes);
  gpuRef = (float *) malloc(nBytes);

  initialData(h_A, nElem);
  initialData(h_B, nElem);

  initialDataInt(R, nElem);
  identityData(I, nElem);

  // Alocando memoria global (GPU)
  float *d_A, *d_B, *d_C;
  CHECK(hipMalloc((float **)&d_A, nBytes));
  CHECK(hipMalloc((float **)&d_B, nBytes));
  CHECK(hipMalloc((float **)&d_C, nBytes));

  // Transferindo dados da CPU pra GPU
  CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
  // CHECK(cudaMemcpy(d_A, h_A, nBytes, cudaMemcpyHostToDevice));

  // Invocando o Kernel na CPU
  int iLen = 512;
  dim3 block(iLen);
  dim3 grid((nElem + block.x - 1) / block.x);
  sumArraysOnGpu<<<grid, block>>>(d_A, d_B, d_C, I, R, strike, nElem);

  // Copia os resultados do Kernel de volta pra CPU
  CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

  // Libera memoria da GPU
  CHECK(hipFree(d_A));
  CHECK(hipFree(d_B));
  CHECK(hipFree(d_C));

  // Libera memória da CPU
  free(h_A);
  free(h_B);
  free(R);
  free(I);


  free(hostRef);
  free(gpuRef);

  hipDeviceReset();
  return 0;
}