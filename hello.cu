#include "hip/hip_runtime.h"
// #include <stdio.h>

__global__ void helloFromGPU() {
  printf("Hello World from GPU! %d %d\n", threadIdx.x, blockIdx.x);
}

int main(int argc, char**argv) {
  printf("Hello World from CPU!\n");
  helloFromGPU<<<4, 6>>>(); CHECK(hipDeviceReset());

  return 0;
}
