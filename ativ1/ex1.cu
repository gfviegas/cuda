
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU() {
  printf("Hello World from GPU! %d %d\n", threadIdx.x, blockIdx.x);
}

int main(int argc, char**argv) {
  printf("Hello World from CPU!\n");

  int blocks = 1;
  int threads = 1024;
  helloFromGPU<<<blocks, threads>>>();

  hipDeviceReset();
  return 0;
}
