
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU() {
  if (threadIdx.x < 20 && blockIdx.x < 20)
    printf("Hello World from GPU! %d %d\n", threadIdx.x, blockIdx.x);
}

int main(int argc, char**argv) {
  printf("Hello World from CPU!\n");

  // 2 milhões blocos de 1024 threads
  long long int blocks = 2 * 1e6;
  long long int threads = 1024; // Numero maximo suportada pela GPU que rodamos
  helloFromGPU<<<blocks, threads>>>();

  hipDeviceReset();
  return 0;
}
