
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU() {
  if (threadIdx.x < 20 && blockIdx.x < 20)
    printf("Hello World from GPU! %d %d\n", threadIdx.x, blockIdx.x);
}

int main(int argc, char**argv) {
  printf("Hello World from CPU!\n");

  // Não tá funcionando colocar esse numero bem grande...
  long long int blocks = 1024;
  long long int threads = 2;
  helloFromGPU<<<blocks, threads>>>();

  hipDeviceReset();
  return 0;
}
