
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void helloFromGPU() {
  printf("Hello World from GPU! %d %d\n", threadIdx.x, blockIdx.x);
}

int main(int argc, char**argv) {
  printf("Hello World from CPU!\n");

  long long int threads = 2 * 1024 * 1024;
  long long int blocks = 1024 * 1024;
  helloFromGPU<<<threads, blocks>>>();

  hipDeviceReset();
  // cudaDeviceSynchronize();
  // CHECK(cudaDeviceReset());

  return 0;
}
