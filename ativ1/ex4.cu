
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

__global__ void fibonacci(int n) {
  int novoN = abs((n - (int) blockIdx.x + (int) threadIdx.x) % n);

  int aux = novoN;
  long long int a = 0;
  long long int b = 1;

  while (aux-- > 1) {
    long long int t = a;
    a = b;
    b += t;
  }

  printf("Fibonacci de %d = %lld\n", novoN, b);
}

int main(int argc, char**argv) {
  printf("Hello World from CPU!\n");

  long long int threads = 1024;
  long long int blocks = 10;
  fibonacci<<<threads, blocks>>>(60);

  hipDeviceSynchronize();
  // cudaDeviceReset();
  // CHECK(cudaDeviceReset());

  return 0;
}
