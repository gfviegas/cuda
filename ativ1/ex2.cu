
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU() {
  printf("Hello World from GPU! %d %d\n", threadIdx.x, blockIdx.x);
}

int main(int argc, char**argv) {
  printf("Hello World from CPU!\n");

  int blocks = 1024;
  int threads = 1;
  helloFromGPU<<<blocks, threads>>>();

  hipDeviceReset();
  return 0;
}
